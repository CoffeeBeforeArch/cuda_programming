#include "hip/hip_runtime.h"
// This program computer the sum of two N-element vectors using unified memory
// By: Nick from CoffeeBeforeArch

#include <stdio.h>
#include <cassert>
#include <iostream>

using std::cout;

// CUDA kernel for vector addition
// No change when using CUDA unified memory
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  // Calculate global thread thread ID
  int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

  // Boundary check
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main() {
  // Array size of 2^16 (65536 elements)
  const int N = 1 << 16;
  size_t bytes = N * sizeof(int);

  // Declare unified memory pointers
  int *a, *b, *c;

  // Allocation memory for these pointers
  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);
  
  // Get the device ID for prefetching calls
  int id = hipGetDevice(&id);

  // Set some hints about the data and do some prefetching
  hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, id);

  // Initialize vectors
  for (int i = 0; i < N; i++) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
  }
  
  // Pre-fetch 'a' and 'b' arrays to the specified device (GPU)
  hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
  hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
  hipMemPrefetchAsync(a, bytes, id);
  hipMemPrefetchAsync(b, bytes, id);
  
  // Threads per CTA (1024 threads per CTA)
  int BLOCK_SIZE = 1 << 10;

  // CTAs per Grid
  int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

  // Call CUDA kernel
  vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

  // Wait for all previous operations before using values
  // We need this because we don't get the implicit synchronization of
  // hipMemcpy like in the original example
  hipDeviceSynchronize();

  // Prefetch to the host (CPU)
  hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

  // Verify the result on the CPU
  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }

  // Free unified memory (same as memory allocated with hipMalloc)
  hipFree(a);
  hipFree(b);
  hipFree(c);

  cout << "COMPLETED SUCCESSFULLY!\n";

  return 0;
}
