// This program computes matrix multiplication using shared memory tiling
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

// Pull out matrix and shared memory tile size
const int M = 1 << 10;
const int N = 1 << 11;
const int K = 1 << 12;
const int SHMEM_SIZE = 1 << 10;

__global__ void matrixMul(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < K; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}

// Check result on the CPU
// MxN = MxK * KxN
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c) {
  // For every row...
  for (int row = 0; row < M; row++) {
    // For every column...
    for (int col = 0; col < N; col++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int i = 0; i < K; i++) {
        // Accumulate the partial results
        tmp += a[row * K + i] * b[i * N + col];
      }

      // Check against the CPU result
      assert(tmp == c[row * N + col]);
    }
  }
}

int main() {
  // Size (in bytes) of matrix
  // MxN = MxK * KxN
  size_t bytes_a = M * K * sizeof(int);
  size_t bytes_b = K * N * sizeof(int);
  size_t bytes_c = M * N * sizeof(int);

  // Host vectors
  vector<int> h_a(M * K);
  vector<int> h_b(K * N);
  vector<int> h_c(M * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes_b, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides M and N evenly)
  int BLOCKS_X = N / THREADS;
  int BLOCKS_Y = M / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS_X, BLOCKS_Y);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  // Check result
  verify_result(h_a, h_b, h_c);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
