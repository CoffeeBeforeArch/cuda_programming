// This program performs sum reduction with an optimization
// removing warp divergence
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <vector>

using std::accumulate;
using std::cout;
using std::generate;
using std::vector;

#define SHMEM_SIZE 256

__global__ void sumReduction(int *v, int *v_r) {
  // Allocate shared memory
  __shared__ int partial_sum[SHMEM_SIZE];

  // Calculate thread ID
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Load elements into shared memory
  partial_sum[threadIdx.x] = v[tid];
  __syncthreads();

  // Increase the stride of the access until we exceed the CTA dimensions
  for (int s = 1; s < blockDim.x; s *= 2) {
    // Change the indexing to be sequential threads
    int index = 2 * s * threadIdx.x;

    // Each thread does work unless the index goes off the block
    if (index < blockDim.x) {
      partial_sum[index] += partial_sum[index + s];
    }
    __syncthreads();
  }

  // Let the thread 0 for this block write it's result to main memory
  // Result is inexed by this block
  if (threadIdx.x == 0) {
    v_r[blockIdx.x] = partial_sum[0];
  }
}

int main() {
  // Vector size
  int N = 1 << 16;
  size_t bytes = N * sizeof(int);

  // Host data
  vector<int> h_v(N);
  vector<int> h_v_r(N);

  // Initialize the input data
  generate(begin(h_v), end(h_v), []() { return rand() % 10; });

  // Allocate device memory
  int *d_v, *d_v_r;
  hipMalloc(&d_v, bytes);
  hipMalloc(&d_v_r, bytes);

  // Copy to device
  hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);

  // TB Size
  const int TB_SIZE = 256;

  // Grid Size (No padding)
  int GRID_SIZE = N / TB_SIZE;

  // Call kernels
  sumReduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r);

  sumReduction<<<1, TB_SIZE>>>(d_v_r, d_v_r);

  // Copy to host;
  hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);

  // Print the result
  assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));

  cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
