// This program computes a sum reduction algortithm with warp divergence
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
#include <numeric>

using std::accumulate;
using std::generate;
using std::cout;
using std::vector;

#define SHMEM_SIZE 256

__global__ void sumReduction(int *v, int *v_r) {
	// Allocate shared memory
	__shared__ int partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements into shared memory
	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	// Iterate of log base 2 the block dimension
	for (int s = 1; s < blockDim.x; s *= 2) {
		// Reduce the threads performing work by half previous the previous
		// iteration each cycle
		if (threadIdx.x % (2 * s) == 0) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

int main() {
	// Vector size
	int N = 1 << 16;
	size_t bytes = N * sizeof(int);

	// Host data
	vector<int> h_v(N);
	vector<int> h_v_r(N);

  // Initialize the input data
  generate(begin(h_v), end(h_v), [](){ return rand() % 10; });

	// Allocate device memory
	int *d_v, *d_v_r;
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);
	
	// Copy to device
	hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);
	
	// TB Size
	const int TB_SIZE = 256;

	// Grid Size (No padding)
	int GRID_SIZE = N / TB_SIZE;

	// Call kernels
	sumReduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r);

	sumReduction<<<1, TB_SIZE>>> (d_v_r, d_v_r);

	// Copy to host;
	hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);

	// Print the result
	assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));

	cout << "COMPLETED SUCCESSFULLY\n";

	return 0;
}
