// This program implements a 1D convolution using CUDA,
// and stores the mask in constant memory. It loads the
// primary array into shared memory, but not halo elements.
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>

// Length of our convolution mask
#define MASK_LENGTH 7

// Allocate space for the mask in constant memory
__constant__ int mask[MASK_LENGTH];

// 1-D convolution kernel
// All threads load 1 element into shared memory
// All threads compute 1 element in final array
//  Arguments:
//      array   = padded array
//      result  = result array
//      n       = number of elements in array
__global__ void convolution_1d(int *array, int *result, int n) {
  // Global thread ID calculation
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Store all elements needed to compute output in shared memory
  extern __shared__ int s_array[];

  // Load elements from the main array into shared memory
  // This is naturally offset by "r" due to padding
  s_array[threadIdx.x] = array[tid];

  __syncthreads();

  // Temp value for calculation
  int temp = 0;

  // Go over each element of the mask
  for (int j = 0; j < MASK_LENGTH; j++) {
    // Get the array value from the caches
    if ((threadIdx.x + j) >= blockDim.x) {
      temp += array[tid + j] * mask[j];
      // Get the value from shared memory
      // Only the last warp will be diverged (given mask size)
    } else {
      temp += s_array[threadIdx.x + j] * mask[j];
    }
  }

  // Write-back the results
  result[tid] = temp;
}

// Verify the result on the CPU
void verify_result(int *array, int *mask, int *result, int n) {
  int temp;
  for (int i = 0; i < n; i++) {
    temp = 0;
    for (int j = 0; j < MASK_LENGTH; j++) {
      temp += array[i + j] * mask[j];
    }
    assert(temp == result[i]);
  }
}

int main() {
  // Number of elements in result array
  int n = 1 << 20;

  // Size of the array in bytes
  int bytes_n = n * sizeof(int);

  // Size of the mask in bytes
  size_t bytes_m = MASK_LENGTH * sizeof(int);

  // Radius for padding the array
  int r = MASK_LENGTH / 2;
  int n_p = n + r * 2;

  // Size of the padded array in bytes
  size_t bytes_p = n_p * sizeof(int);

  // Allocate the array (include edge elements)...
  int *h_array = new int[n_p];

  // ... and initialize it
  for (int i = 0; i < n_p; i++) {
    if ((i < r) || (i >= (n + r))) {
      h_array[i] = 0;
    } else {
      h_array[i] = rand() % 100;
    }
  }

  // Allocate the mask and initialize it
  int *h_mask = new int[MASK_LENGTH];
  for (int i = 0; i < MASK_LENGTH; i++) {
    h_mask[i] = rand() % 10;
  }

  // Allocate space for the result
  int *h_result = new int[n];

  // Allocate space on the device
  int *d_array, *d_result;
  hipMalloc(&d_array, bytes_p);
  hipMalloc(&d_result, bytes_n);

  // Copy the data to the device
  hipMemcpy(d_array, h_array, bytes_p, hipMemcpyHostToDevice);

  // Copy the mask directly to the symbol
  // This would require 2 API calls with cudaMemcpy
  hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

  // Threads per TB
  int THREADS = 256;

  // Number of TBs
  int GRID = (n + THREADS - 1) / THREADS;

  // Amount of space per-block for shared memory
  // This is padded by the overhanging radius on either side
  size_t SHMEM = THREADS * sizeof(int);

  // Call the kernel
  convolution_1d<<<GRID, THREADS, SHMEM>>>(d_array, d_result, n);

  // Copy back the result
  hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

  // Verify the result
  verify_result(h_array, h_mask, h_result, n);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free allocated memory on the device and host
  delete[] h_array;
  delete[] h_result;
  delete[] h_mask;
  hipFree(d_array);
  hipFree(d_result);

  return 0;
}
