#include "hip/hip_runtime.h"
// This program calculates matrix multiplication (SGEMM) using cuBLAS
// By: Nick from CoffeeBeforeArch

#include <hipblas.h>
#include <hiprand.h>
#include <cassert>
#include <cmath>
#include <ctime>
#include <iostream>
#include <vector>

// Verify our result on the CPU
void verify_solution(float *a, float *b, float *c, int n) {
  float epsilon = 0.001;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      float temp = 0;
      for (int k = 0; k < n; k++) {
        temp += a[k * n + i] * b[j * n + k];
      }
      assert(fabs(c[j * n + i] - temp) < epsilon);
    }
  }
}

int main() {
  // Dimensions for our matrices
  // MxN * NxK = MxK
  int M = 1 << 8;
  int N = 1 << 9;
  int K = 1 << 10;

  // Pre-calculate the size (in bytes) of our matrices
  size_t bytes_a = M * N * sizeof(float);
  size_t bytes_b = N * K * sizeof(float);
  size_t bytes_c = M * K * sizeof(float);

  // Vectors for the host data
  std::vector<float> h_a(M * N);
  std::vector<float> h_b(N * K);
  std::vector<float> h_c(M * K);

  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Pseudo random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the matrix with random numbers on the device
  hiprandGenerateUniform(prng, d_a, M * N);
  hiprandGenerateUniform(prng, d_b, N * K);

  // cuBLAS handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Scalaing factors
  float alpha = 1.0f;
  float beta = 0.0f;

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxK = MxN * NxK
  // Signature: handle, operation, operation, m, n, k, alpha, A, lda, B, ldb,
  // beta, C, ldc
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, N,
              &beta, d_c, M);

  // Copy back the three matrices
  hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  // Verify solution
  verify_solution(h_a, h_b, h_c, n);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free our memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
