// This program calculates matrix multiplication (SGEMM) using cuBLAS
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cassert>
#include <cmath>
#include <ctime>
#include <iostream>
#include <vector>

// Verify our result on the CPU
void verify_solution(float *a, float *b, float *c, int M, int N, int K) {
  // Tolerance for our result (floats are imperfect)
  float epsilon = 0.001f;

  // For every row...
  for (int row = 0; row < M; row++) {
    // For every column
    for (int col = 0; col < N; col++) {
      // For every element in the row-col pair...
      float temp = 0;
      for (int i = 0; i < K; i++) {
        temp += a[row + M * i] * b[col * K + i];
      }
      assert(fabs(c[col * M + row] - temp) <= epsilon);
    }
  }
}

int main() {
  // Dimensions for our matrices
  // MxK * KxN = MxN
  const int M = 1 << 9;
  const int N = 1 << 8;
  const int K = 1 << 7;

  // Pre-calculate the size (in bytes) of our matrices
  const size_t bytes_a = M * K * sizeof(float);
  const size_t bytes_b = K * N * sizeof(float);
  const size_t bytes_c = M * N * sizeof(float);

  // Vectors for the host data
  std::vector<float> h_a(M * K);
  std::vector<float> h_b(K * N);
  std::vector<float> h_c(M * N);
  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);
  hipMemcpy(d_c, h_c.data(), bytes_c, hipMemcpyHostToDevice);

  // Pseudo random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the matrix with random numbers on the device
  hiprandGenerateUniform(prng, d_a, M * K);
  hiprandGenerateUniform(prng, d_b, K * M);

  // cuBLAS handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Scalaing factors
  float alpha = 1.0f;
  float beta = 0.0f;

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxN = MxK * KxN
  // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
  // beta, C, ldc
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, K,
              &beta, d_c, M);

  // Copy back the three matrices
  hipMemcpy(h_a.data(), d_a, bytes_a, hipMemcpyDeviceToHost);
  hipMemcpy(h_b.data(), d_b, bytes_b, hipMemcpyDeviceToHost);
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  // Verify solution
  verify_solution(h_a.data(), h_b.data(), h_c.data(), M, N, K);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free our memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
