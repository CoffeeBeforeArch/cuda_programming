// This program calculates matrix multiplication (SGEMM) using cuBLAS
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cassert>
#include <cmath>
#include <ctime>
#include <iostream>
#include <vector>

// Verify our result on the CPU
void verify_solution(float *a, float *b, float *c, int M, int N, int K) {
  float epsilon = 0.001;
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < K; j++) {
      float temp = 0;
      for (int k = 0; k < N; k++) {
        temp += a[k * N + i] * b[j * K + k];
      }
      assert(fabs(c[j * N + i] - temp) < epsilon);
    }
  }
}

int main() {
  // Dimensions for our matrices
  // MxN * NxK = MxK
  const int M = 1 << 8;
  const int N = 1 << 9;
  const int K = 1 << 7;

  // Pre-calculate the size (in bytes) of our matrices
  const size_t bytes_a = M * N * sizeof(float);
  const size_t bytes_b = N * K * sizeof(float);
  const size_t bytes_c = M * K * sizeof(float);

  // Vectors for the host data
  std::vector<float> h_a(M * N);
  std::vector<float> h_b(N * K);
  std::vector<float> h_c(M * K);

  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // Pseudo random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the matrix with random numbers on the device
  hiprandGenerateUniform(prng, d_a, M * N);
  hiprandGenerateUniform(prng, d_b, N * K);

  // cuBLAS handle
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Scalaing factors
  float alpha = 1.0f;
  float beta = 0.0f;

  // Calculate: c = (alpha*a) * b + (beta*c)
  // MxK = MxN * NxK
  // Signature: handle, operation, operation, m, n, k, alpha, A, lda, B, ldb,
  // beta, C, ldc
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, N,
              &beta, d_c, M);

  // Copy back the three matrices
  hipMemcpy(h_a.data(), d_a, bytes_a, hipMemcpyDeviceToHost);
  hipMemcpy(h_b.data(), d_b, bytes_b, hipMemcpyDeviceToHost);
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);

  // Verify solution
  verify_solution(h_a.data(), h_b.data(), h_c.data(), M, N, K);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free our memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
