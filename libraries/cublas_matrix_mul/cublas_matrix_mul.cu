// This program calculates matrix multiplication (SGEMM) using cuBLAS
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

void verify_solution(float *a, float *b, float *c, int n) {
	float temp;
	float epsilon = 0.001;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			temp = 0;
			for (int k = 0; k < n; k++) {
				temp += a[k * n + i] * b[j * n + k];
			}
			assert(fabs(c[j * n + i] - temp) < epsilon);
		}
	}
}

int main() {
	// Problem size
	int n = 1 << 10;
	size_t bytes = n * n * sizeof(float);

	// Declare pointers to matrices on device and host
	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

	// Allocate memory
	h_a = (float*)malloc(bytes);
	h_b = (float*)malloc(bytes);
	h_c = (float*)malloc(bytes);
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Pseudo random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

	// Fill the matrix with random numbers on the device
	hiprandGenerateUniform(prng, d_a, n*n);
	hiprandGenerateUniform(prng, d_b, n*n);

	// cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Scalaing factors
	float alpha = 1.0f;
	float beta = 0.0f;

	// Calculate: c = (alpha*a) * b + (beta*c)
	// (m X n) * (n X k) = (m X k)
	// Signature: handle, operation, operation, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

	// Copy back the three matrices
	hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Verify solution
	verify_solution(h_a, h_b, h_c, n);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}