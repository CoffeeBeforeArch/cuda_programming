// This program implements a 1D convolution using CUDA
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

// 1-D convolution kernel
//  Arguments:
//      array   = padded array
//      mask    = convolution mask
//      result  = result array
//      n       = number of elements in array
//      m       = number of elements in the mask
__global__ void convolution_1d(int *array, int *mask, int *result, int n,
                               int m) {
  // Global thread ID calculation
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Calculate radius of the mask
  int r = m / 2;

  // Calculate the starting point for the element
  int start = tid - r;

  // Temp value for calculation
  int temp = 0;

  // Go over each element of the mask
  for (int j = 0; j < m; j++) {
    // Ignore elements that hang off (0s don't contribute)
    if (((start + j) >= 0) && (start + j < n)) {
      // accumulate partial results
      temp += array[start + j] * mask[j];
    }
  }

  // Write-back the results
  result[tid] = temp;
}

// Verify the result on the CPU
void verify_result(int *array, int *mask, int *result, int n, int m) {
  int radius = m / 2;
  int temp;
  int start;
  for (int i = 0; i < n; i++) {
    start = i - radius;
    temp = 0;
    for (int j = 0; j < m; j++) {
      if ((start + j >= 0) && (start + j < n)) {
        temp += array[start + j] * mask[j];
      }
    }
    assert(temp == result[i]);
  }
}

int main() {
  // Number of elements in result array
  int n = 1 << 20;

  // Size of the array in bytes
  int bytes_n = n * sizeof(int);

  // Number of elements in the convolution mask
  int m = 7;

  // Size of mask in bytes
  int bytes_m = m * sizeof(int);

  // Allocate the array (include edge elements)...
  std::vector<int> h_array(n);

  // ... and initialize it
  std::generate(begin(h_array), end(h_array), [](){ return rand() % 100; });

  // Allocate the mask and initialize it
  std::vector<int> h_mask(m);
  std::generate(begin(h_mask), end(h_mask), [](){ return rand() % 10; });

  // Allocate space for the result
  std::vector<int> h_result(n);

  // Allocate space on the device
  int *d_array, *d_mask, *d_result;
  hipMalloc(&d_array, bytes_n);
  hipMalloc(&d_mask, bytes_m);
  hipMalloc(&d_result, bytes_n);

  // Copy the data to the device
  hipMemcpy(d_array, h_array.data(), bytes_n, hipMemcpyHostToDevice);
  hipMemcpy(d_mask, h_mask.data(), bytes_m, hipMemcpyHostToDevice);

  // Threads per TB
  int THREADS = 256;

  // Number of TBs
  int GRID = (n + THREADS - 1) / THREADS;

  // Call the kernel
  convolution_1d<<<GRID, THREADS>>>(d_array, d_mask, d_result, n, m);

  // Copy back the result
  hipMemcpy(h_result.data(), d_result, bytes_n, hipMemcpyDeviceToHost);

  // Verify the result
  verify_result(h_array.data(), h_mask.data(), h_result.data(), n, m);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free allocated memory on the device and host
  hipFree(d_result);
  hipFree(d_mask);
  hipFree(d_array);

  return 0;
}
