// This program implements a 1D convolution using CUDA,
// and stores the mask in constant memory
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>

// Length of our convolution mask
#define MASK_LENGTH 7

// Allocate space for the mask in constant memory
__constant__ int mask[MASK_LENGTH];

// 1-D convolution kernel
//  Arguments:
//      array   = padded array
//      result  = result array
//      n       = number of elements in array
__global__ void convolution_1d(int *array, int *result, int n) {
  // Global thread ID calculation
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Calculate radius of the mask
  int r = MASK_LENGTH / 2;

  // Calculate the starting point for the element
  int start = tid - r;

  // Temp value for calculation
  int temp = 0;

  // Go over each element of the mask
  for (int j = 0; j < MASK_LENGTH; j++) {
    // Ignore elements that hang off (0s don't contribute)
    if (((start + j) >= 0) && (start + j < n)) {
      // accumulate partial results
      temp += array[start + j] * mask[j];
    }
  }

  // Write-back the results
  result[tid] = temp;
}

// Verify the result on the CPU
void verify_result(int *array, int *mask, int *result, int n) {
  int radius = MASK_LENGTH / 2;
  int temp;
  int start;
  for (int i = 0; i < n; i++) {
    start = i - radius;
    temp = 0;
    for (int j = 0; j < MASK_LENGTH; j++) {
      if ((start + j >= 0) && (start + j < n)) {
        temp += array[start + j] * mask[j];
      }
    }
    assert(temp == result[i]);
  }
}

int main() {
  // Number of elements in result array
  int n = 1 << 20;

  // Size of the array in bytes
  int bytes_n = n * sizeof(int);

  // Size of the mask in bytes
  size_t bytes_m = MASK_LENGTH * sizeof(int);

  // Allocate the array (include edge elements)...
  int *h_array = new int[n];

  // ... and initialize it
  for (int i = 0; i < n; i++) {
    h_array[i] = rand() % 100;
  }

  // Allocate the mask and initialize it
  int *h_mask = new int[MASK_LENGTH];
  for (int i = 0; i < MASK_LENGTH; i++) {
    h_mask[i] = rand() % 10;
  }

  // Allocate space for the result
  int *h_result = new int[n];

  // Allocate space on the device
  int *d_array, *d_result;
  hipMalloc(&d_array, bytes_n);
  hipMalloc(&d_result, bytes_n);

  // Copy the data to the device
  hipMemcpy(d_array, h_array, bytes_n, hipMemcpyHostToDevice);

  // Copy the data directly to the symbol
  // Would require 2 API calls with cudaMemcpy
  hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

  // Threads per TB
  int THREADS = 256;

  // Number of TBs
  int GRID = (n + THREADS - 1) / THREADS;

  // Call the kernel
  convolution_1d<<<GRID, THREADS>>>(d_array, d_result, n);

  // Copy back the result
  hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

  // Verify the result
  verify_result(h_array, h_mask, h_result, n);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free allocated memory on the device and host
  delete[] h_array;
  delete[] h_result;
  delete[] h_mask;
  hipFree(d_result);

  return 0;
}
