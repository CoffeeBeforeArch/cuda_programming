// This program performs sum reduction with an optimization
// removing warp bank conflicts
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#define SIZE 256
#define SHMEM_SIZE 256 * 4

__global__ void sum_reduction(int *v, int *v_r) {
	// Allocate shared memory
	__shared__ int partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements into shared memory
	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

void initialize_vector(int *v, int n) {
	for (int i = 0; i < n; i++) {
		v[i] = 1;//rand() % 10;
	}
}

int main() {
	// Vector size
	int n = 1 << 16;
	size_t bytes = n * sizeof(int);

	// Original vector and result vector
	int *h_v, *h_v_r;
	int *d_v, *d_v_r;

	// Allocate memory
	h_v = (int*)malloc(bytes);
	h_v_r = (int*)malloc(bytes);
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	// Initialize vector
	initialize_vector(h_v, n);

	// Copy to device
	hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice);

	// TB Size
	int TB_SIZE = SIZE;

	// Grid Size (No padding)
	int GRID_SIZE = n / TB_SIZE;

	// Call kernel
	sum_reduction <<<GRID_SIZE, TB_SIZE >>> (d_v, d_v_r);

	sum_reduction <<<1, TB_SIZE >>> (d_v_r, d_v_r);

	// Copy to host;
	hipMemcpy(h_v_r, d_v_r, bytes, hipMemcpyDeviceToHost);

	// Print the result
	//printf("Accumulated result is %d \n", h_v_r[0]);
	//scanf("Press enter to continue: ");
	assert(h_v_r[0] == 65536);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}
